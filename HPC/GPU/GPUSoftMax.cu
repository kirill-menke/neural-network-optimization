#include "hip/hip_runtime.h"
#include <limits>
#include <cmath>

#include "GPUSoftMax.h"
#include "cuda-utils.h"

__global__
static void forwardKernel(Tensor<float, 4> input, Tensor<float, 4> output)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int b = threadIdx.z + blockIdx.z * blockDim.z;

	int batch_size = input.dim(0), channels = input.dim(1), width = input.dim(2), height = input.dim(3);
	if (b >= batch_size || x >= width || y >= height)
		return;

	float max = -std::numeric_limits<float>::infinity();
	for (int c = 0; c < channels; c++) {
		float val = input(b, c, x, y);
		if (val > max)
			max = val;
	}

	float sum = 0.;
	for (int c = 0; c < channels; c++) {
		float val = expf(input(b, c, x, y) - max);
		output(b, c, x, y) = val;
		sum += val;
	}

	for (int c = 0; c < channels; c++)
		output(b, c, x, y) /= sum;
}

__global__
static void backwardKernel(Tensor<float, 4> error, Tensor<float, 4> output, Tensor<float, 4> next_error)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int b = threadIdx.z + blockIdx.z * blockDim.z;

	int batch_size = error.dim(0), channels = error.dim(1), width = error.dim(2), height = error.dim(3);
	if (b >= batch_size || x >= width || y >= height)
		return;

	float sum = 0.;
	for (int c = 0; c < channels; c++)
		sum += error(b, c, x, y) * output(b, c, x, y);

	for (int c = 0; c < channels; c++)
		next_error(b, c, x, y) = output(b, c, x, y) * (error(b, c, x, y) - sum);
}

Tensor<float, 4> GPUSoftMax::forward(Tensor<float, 4> &input) {
	this->output_tensor = new Tensor<float, 4>({
		input.dim(0), input.dim(1), input.dim(2), input.dim(3) });

	dim3 gridDim = getGridDim(input.dim(2), input.dim(3), input.dim(0));
	dim3 blockDim = getBlockDim(input.dim(2), input.dim(3), input.dim(0));

	forwardKernel<<<gridDim, blockDim>>>(input, *output_tensor);
	return *output_tensor;
}

Tensor<float, 4> GPUSoftMax::backward(Tensor<float, 4> &error) {
	Tensor<float, 4> next_error({
		error.dim(0), error.dim(1), error.dim(2), error.dim(3) });

	dim3 gridDim = getGridDim(error.dim(2), error.dim(3), error.dim(0));
	dim3 blockDim = getBlockDim(error.dim(2), error.dim(3), error.dim(0));

	backwardKernel<<<gridDim, blockDim>>>(error, *(this->output_tensor), next_error);
	return next_error;
}

