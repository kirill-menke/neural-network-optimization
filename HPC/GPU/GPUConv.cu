#include "hip/hip_runtime.h"
#include <assert.h>

#include "./GPUConv.h"
#include "./cuda-utils.h"

/*
 * Call this kernel per output pixel X/Y and batch.
 */
__global__
static void addPadding(Tensor<float, 4> input, Tensor<float, 4> output, int px, int py) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int b = threadIdx.z + blockIdx.z * blockDim.z;

	int batchSize = input.dim(0), channels = input.dim(1),
	    inputWidth = input.dim(2), inputHeight = input.dim(3),
	    outputWidth = output.dim(2), outputHeight = output.dim(3);

	if (x >= outputWidth || y >= outputHeight)
		return;

	for (int c = 0; c < channels; c++) {
		if (x < px || y < py || x >= px + inputWidth || y >= py + inputHeight)
			output(b, c, x, y) = 0.0;
		else
			output(b, c, x, y) = input(b, c, x - px, y - py);
	}
}

/*
 * Upsample (used for error_tensor in backward)
 *
 * Call this kernel per output pixel X/Y and batch.
 */
__global__
static void upsample(Tensor<float, 4> input, Tensor<float, 4> output, int strideX, int strideY) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int b = threadIdx.z + blockIdx.z * blockDim.z;

	int batchSize = input.dim(0), channels = input.dim(1),
	    inputWidth = input.dim(2), inputHeight = input.dim(3),
	    outputWidth = output.dim(2), outputHeight = output.dim(3);

	if (x >= outputWidth || y >= outputHeight || b >= batchSize)
		return;

	for (int c = 0; c < channels; c++) {
		if (x % strideX == 0 && y % strideY == 0 && x / strideX < inputWidth && y / strideY < inputHeight)
			output(b, c, x, y) = input(b, c, x / strideX, y / strideY);
		else
			output(b, c, x, y) = 0.0;
	}
}

/*
 * Optimizations:
 * - TODO: Copy filters into shared memory
 * - TODO: Not sure: Copy input into shared memory?
 *
 * Call this kernel per output pixel X/Y and batch.
 */
template<bool backward>
__global__
static void convolution(Tensor<float, 4> input, Tensor<float, 4> output, Tensor<float, 4> weights, Tensor<float, 1> bias, int strideX, int strideY) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int b = threadIdx.z + blockIdx.z * blockDim.z;

	int batchSize = input.dim(0), inputChannels = input.dim(1),
	    outputWidth = output.dim(2), outputHeight = output.dim(3),
	    outputChannels = output.dim(1);

	int filterWidth = weights.dim(2), filterHeight = weights.dim(3);

	if (b > batchSize || x >= outputWidth || y >= outputHeight)
		return;

	for (int cout = 0; cout < outputChannels; cout++) {
		float value = 0;

		for (int cin = 0; cin < inputChannels; cin++) {
			for (int i = 0; i < filterWidth; i++) {
				for (int j = 0; j < filterHeight; j++) {
					float inputVal = input(b, cin, x * strideX + i, y * strideY + j);
					if (backward)
						value += inputVal * weights.flipped(cin, cout, i, j);
					else
						value += inputVal * weights(cout, cin, i, j);
				}
			}
		}

		if (!backward)
			value += bias(cout);

		output(b, cout, x, y) = value;
	}
}

/*
 * Calculate gradient_weights:
 *
 * Call this kernel per filter pixel X/Y and batch (TODO: Could also include filter here for perf.)
 * TODO: Can this be represented as a wired convolution as well?
 */
__global__
void convBackwardGradientWeights(Tensor<float, 4> error_tensor, Tensor<float, 4> input_tensor, Tensor<float, 4> gradient_weights, int strideX, int strideY) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int b = threadIdx.z + blockIdx.z * blockDim.z;

	int filterWidth = gradient_weights.dim(2), filterHeight = gradient_weights.dim(3);
	int batchSize = input_tensor.dim(0);
	if (i >= filterWidth || j >= filterHeight || b >= batchSize)
		return;

	int imageWidth = error_tensor.dim(2), imageHeight = error_tensor.dim(3);
	int inputChannels = input_tensor.dim(1), outputChannels = error_tensor.dim(1);
	for (int f = 0; f < outputChannels; f++) {
		for (int c = 0; c < inputChannels; c++) {
			float err = 0.0;

			for (int x = 0; x < imageWidth; x += strideX) {
				for (int y = 0; y < imageHeight; y += strideY) {
					err += input_tensor(b, c, x + i, y + j) * error_tensor(b, f, x, y);
				}
			}

			gradient_weights(f, c, i, j) = err;
		}
	}
}

/*
 * TODO: Do some sort of parallel reduction instead?
 */
__global__
void convBackwardGradientBias(Tensor<float, 4> error_tensor, Tensor<float, 1> gradient_bias) {
	int f = threadIdx.x + blockIdx.x * blockDim.x;
	if (f >= gradient_bias.dim(0))
		return;

	int batchSize = error_tensor.dim(0),
	    width = error_tensor.dim(2),
	    height = error_tensor.dim(3);

	float value = 0.;
	for (int b = 0; b < batchSize; b++) {
		for (int x = 0; x < width; x++) {
			for (int y = 0; y < height; y++) {
				value += error_tensor(b, f, x, y);
			}
		}
	}
	gradient_bias(f) = value;
}

Tensor<float, 4> GPUConv::forward(Tensor<float, 4> &input_tensor) {
	int batchSize = input_tensor.dim(0);

	assert(input_tensor.dim(1) == inputChannels);
	assert(input_tensor.dim(2) == imageWidth);
	assert(input_tensor.dim(3) == imageHeight);

	this->padded_input = new Tensor<float, 4>(Tensor<float, 4>::ON_GPU, {
		batchSize,
		inputChannels,
		imageWidth + 2 * (filterWidth / 2),
		imageHeight + 2 * (filterHeight / 2)
	});

	{
		dim3 gridDim = getGridDim(padded_input->dim(2), padded_input->dim(3), batchSize);
		dim3 blockDim = getBlockDim(padded_input->dim(2), padded_input->dim(3), batchSize);
		addPadding<<<gridDim, blockDim>>>(input_tensor, *padded_input, filterWidth / 2, filterHeight / 2);
	}

	int outputWidth = imageWidth / strideX, outputHeight = imageHeight / strideY;

	Tensor<float, 4> output_tensor(Tensor<float, 4>::ON_GPU, {
		batchSize,
		outputChannels,
		outputWidth,
		outputHeight
	});

	{
		dim3 gridDim = getGridDim(outputWidth, outputHeight, batchSize);
		dim3 blockDim = getBlockDim(outputWidth, outputHeight, batchSize);
		convolution<false><<<gridDim, blockDim>>>(*padded_input, output_tensor, weights, bias, strideX, strideY);
	}

	return output_tensor;
}

/*
 * Opt./Simplify for stride = 1?
 */
Tensor<float, 4> GPUConv::backward(Tensor<float, 4> &error_tensor) {
	int batchSize = error_tensor.dim(0);
	int outputWidth = imageWidth / strideX, outputHeight = imageHeight / strideY;
	assert(error_tensor.dim(1) == inputChannels);
	assert(error_tensor.dim(2) == outputWidth);
	assert(error_tensor.dim(3) == outputHeight);
	assert(padded_input->dim(0) == batchSize);

	Tensor<float, 4> upsampled_error_tensor(Tensor<float, 4>::ON_GPU, {
		batchSize,
		outputChannels,
		imageWidth,
		imageHeight
	});

	{
		dim3 gridDim = getGridDim(imageWidth, imageHeight, batchSize);
		dim3 blockDim = getBlockDim(imageWidth, imageHeight, batchSize);
		upsample<<<gridDim, blockDim>>>(error_tensor, upsampled_error_tensor, strideX, strideY);
	}

	Tensor<float, 4> padded_error_tensor(Tensor<float, 4>::ON_GPU, {
		batchSize,
		outputChannels,
		imageWidth + 2 * (filterWidth / 2),
		imageHeight + 2 * (filterHeight / 2)
	});

	{
		dim3 gridDim = getGridDim(padded_error_tensor.dim(2), padded_error_tensor.dim(3), batchSize);
		dim3 blockDim = getBlockDim(padded_error_tensor.dim(2), padded_error_tensor.dim(3), batchSize);
		addPadding<<<gridDim, blockDim>>>(upsampled_error_tensor, padded_error_tensor, filterWidth / 2, filterHeight / 2);
	}


	Tensor<float, 4> next_error_tensor(Tensor<float, 4>::ON_GPU, {
		batchSize,
		inputChannels,
		imageWidth,
		imageHeight
	});

	{
		dim3 gridDim = getGridDim(imageWidth, imageHeight, batchSize);
		dim3 blockDim = getBlockDim(imageWidth, imageHeight, batchSize);
		convolution<true><<<gridDim, blockDim>>>(padded_error_tensor, next_error_tensor, weights, bias, 1, 1);
	}

	Tensor<float, 4> gradient_weights(Tensor<float, 4>::ON_GPU, {
		outputChannels,
		inputChannels,
		filterWidth,
		filterHeight
	});

	{
		dim3 gridDim = getGridDim(filterWidth, filterHeight, batchSize);
		dim3 blockDim = getBlockDim(filterWidth, filterHeight, batchSize);
		convBackwardGradientWeights<<<gridDim, blockDim>>>(upsampled_error_tensor, *padded_input, gradient_weights, strideX, strideY);
	}

	Tensor<float, 1> gradient_bias(Tensor<float, 1>::ON_GPU, { outputChannels });

	{
		dim3 gridDim = getGridDim(outputChannels, 1, 1);
		dim3 blockDim = getBlockDim(outputChannels, 1, 1);
		convBackwardGradientBias<<<gridDim, blockDim>>>(error_tensor, gradient_bias);
	}

	// TODO: Pass gradient_weights/bias on to Optimizer or something like that!
	gradient_weights.destroy();
	gradient_bias.destroy();

	padded_error_tensor.destroy();
	upsampled_error_tensor.destroy();

	this->padded_input->destroy();
	delete this->padded_input;
	this->padded_input = nullptr;
	return next_error_tensor;
}
